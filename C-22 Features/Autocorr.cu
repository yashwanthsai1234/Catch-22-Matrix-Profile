#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <cmath>

__global__ void mean_kernel(const double* d_input, double* d_partial_sums, int size) {
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = (i < size) ? d_input[i] : 0;
    __syncthreads();
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) d_partial_sums[blockIdx.x] = sdata[0];
}

double compute_mean_cuda(const double* h_input, int size) {
    double *d_input, *d_partial_sums, *h_partial_sums;
    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    int partialSumsSize = blocks * sizeof(double);

    hipError_t cudaStatus = hipMalloc(&d_input, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_input: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }

    cudaStatus = hipMalloc(&d_partial_sums, partialSumsSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_partial_sums: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input);
        return 0;
    }

    h_partial_sums = (double*)malloc(partialSumsSize);
    if (!h_partial_sums) {
        fprintf(stderr, "Failed to allocate host memory for h_partial_sums\n");
        hipFree(d_input);
        hipFree(d_partial_sums);
        return 0;
    }

    cudaStatus = hipMemcpy(d_input, h_input, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input);
        hipFree(d_partial_sums);
        free(h_partial_sums);
        return 0;
    }

    mean_kernel<<<blocks, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(d_input, d_partial_sums, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mean_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input);
        hipFree(d_partial_sums);
        free(h_partial_sums);
        return 0;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mean_kernel: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        hipFree(d_input);
        hipFree(d_partial_sums);
        free(h_partial_sums);
        return 0;
    }

    cudaStatus = hipMemcpy(h_partial_sums, d_partial_sums, partialSumsSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_input);
        hipFree(d_partial_sums);
        free(h_partial_sums);         
        return 0;
    }

    double totalSum = 0;
    for (int i = 0; i < blocks; i++) {
        totalSum += h_partial_sums[i];
    }

    hipFree(d_input);
    hipFree(d_partial_sums);
    free(h_partial_sums);
    return totalSum / size;
}

__global__ void subtract_mean_and_pad(double *d_y, int size, int nFFT, double mean) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_y[idx] -= mean;
    }
    else if (idx < nFFT) {
        d_y[idx] = 0.0;
    }
}

__global__ void complex_conjugate_multiply(hipfftDoubleComplex *data, int nFFT) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nFFT) {
        hipfftDoubleComplex val = data[idx];
        hipfftDoubleComplex conjVal = hipConj(val);
        data[idx] = hipCmul(val, conjVal);
    }
}

__device__ double cufftComplex_abs(hipfftDoubleComplex z) {
    return sqrt(z.x * z.x + z.y * z.y);
}

int nextpow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void normalize(double *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] /= data[0];
    }
}

double *cuda_co_autocorrs(const double *y, const int size) {
    int nFFT =  nextpow2(size);
    double *d_y, *autocorr;
    hipfftDoubleComplex *d_freqDomain;
    hipfftHandle plan_f, plan_i;
    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    autocorr = (double *)malloc(nFFT * sizeof(double));
    if (!autocorr) {
        fprintf(stderr, "Failed to allocate host memory for autocorr\n");
        return NULL;
    }

    cudaStatus = hipMalloc(&d_y, nFFT * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_y: %s\n", hipGetErrorString(cudaStatus));
        free(autocorr);
        return NULL;
    }

    cudaStatus = hipMalloc(&d_freqDomain, nFFT * sizeof(hipfftDoubleComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_freqDomain: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_y);
        free(autocorr);
        return NULL;
    }

    cudaStatus = hipMemcpy(d_y, y, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_y);
        hipFree(d_freqDomain);
        free(autocorr);
        return NULL;
    }

    double *zeroPad = (double *)calloc(nFFT - size, sizeof(double));
    hipMemcpy(d_y + size, zeroPad, (nFFT - size) * sizeof(double), hipMemcpyHostToDevice);
    free(zeroPad);

    cufftStatus = hipfftPlan1d(&plan_f, nFFT, HIPFFT_D2Z, 1);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed with error code %d\n", cufftStatus);
        hipFree(d_y);
        hipFree(d_freqDomain);
        free(autocorr);
        return NULL;
    }

    cufftStatus = hipfftPlan1d(&plan_i, nFFT, HIPFFT_Z2D, 1);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed with error code %d\n", cufftStatus);
        hipFree(d_y);
        hipFree(d_freqDomain);
        free(autocorr);
        return NULL;
    }

    cufftStatus = hipfftExecD2Z(plan_f, d_y, d_freqDomain);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecD2Z failed with error code %d\n", cufftStatus);
        hipFree(d_y);
        hipFree(d_freqDomain);
        free(autocorr);
        return NULL;
    }

    int threadsPerBlock = 256;
    int blocks = (nFFT + threadsPerBlock - 1) / threadsPerBlock;
    complex_conjugate_multiply<<<blocks, threadsPerBlock>>>(d_freqDomain, nFFT);
    hipDeviceSynchronize();

    cufftStatus = hipfftExecZ2D(plan_i, d_freqDomain, d_y);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecZ2D failed with error code %d\n", cufftStatus);
        hipfftDestroy(plan_i);
        free(autocorr);
        return NULL;
    }

    int blocksPerGrid = (nFFT + threadsPerBlock - 1) / threadsPerBlock;
    normalize<<<blocksPerGrid, threadsPerBlock>>>(d_y, nFFT);
    hipDeviceSynchronize();

    cudaStatus = hipMemcpy(autocorr, d_y, nFFT * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_y);
        hipFree(d_freqDomain);
        free(autocorr);
        return NULL;
    }

    hipfftDestroy(plan_f);
    hipfftDestroy(plan_i);
    hipFree(d_y);
    hipFree(d_freqDomain);
    return autocorr;
}

__global__ void find_first_min_kernel(const double *autocorrs, int size, int *minIndex) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < size - 1 && autocorrs[i] < autocorrs[i - 1] && autocorrs[i] < autocorrs[i + 1]) {
        atomicMin(minIndex, i);
    }
}

int CO_FirstMin_ac_cuda(const double y[], const int size) {
    double *autocorrs = cuda_co_autocorrs(y, size);
    double *d_autocorrs;
    int *d_minIndex;
    int h_minIndex = size;
    hipMalloc(&d_autocorrs, size * sizeof(double));
    hipMalloc(&d_minIndex, sizeof(int));
    hipMemcpy(d_autocorrs, autocorrs, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_minIndex, &h_minIndex, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    find_first_min_kernel<<<blocks, threadsPerBlock>>>(d_autocorrs, size, d_minIndex);

    hipMemcpy(&h_minIndex, d_minIndex, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_autocorrs);
    hipFree(d_minIndex);
    free(autocorrs);
    return h_minIndex;
}

__global__ void findThresholdCrossing(const double* autocorr, int size, double thresh, int* crossingIndex) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > 0 && idx < size) {
        if (autocorr[idx] < thresh && autocorr[idx - 1] >= thresh) {
            atomicMin(crossingIndex, idx);
        }
    }
}

double CO_f1ecac_CUDA(const double* y, int size) {
    double* autocorr_d = nullptr;
    int* crossingIndex_d = nullptr;
    int crossingIndex_h = INT_MAX;
    hipMalloc((void**)&autocorr_d, size * sizeof(double));
    hipMemcpy(autocorr_d, y, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void**)&crossingIndex_d, sizeof(int));
    hipMemcpy(crossingIndex_d, &crossingIndex_h, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    double thresh = 1.0 / exp(1);
    findThresholdCrossing<<<blocksPerGrid, threadsPerBlock>>>(autocorr_d, size, thresh, crossingIndex_d);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findThresholdCrossing!\n", cudaStatus);
    }

    hipMemcpy(&crossingIndex_h, crossingIndex_d, sizeof(int), hipMemcpyDeviceToHost);
    double out = (double)size;
    if (crossingIndex_h != INT_MAX && crossingIndex_h > 0 && crossingIndex_h < size) {
        double autocorr_values[2];
        hipMemcpy(autocorr_values, &autocorr_d[crossingIndex_h - 1], 2 * sizeof(double), hipMemcpyDeviceToHost);
        double m = autocorr_values[1] - autocorr_values[0];
        double dy = thresh - autocorr_values[0];
        double dx = dy / m;
        out = crossingIndex_h - 1 + dx;
    } else {
        printf("Threshold crossing not found.\n");
    }

    hipFree(autocorr_d);
    hipFree(crossingIndex_d);
    return out;
}

__global__ void compute_cubed_differences(const double *y, double *cubed_diffs, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size - 1) {
        double diff = y[idx + 1] - y[idx];
        cubed_diffs[idx] = diff * diff * diff;
    }
}

double CO_trev_1_num_cuda(const double *y, int size) {
    double *d_y, *d_cubed_diffs;
    hipMalloc(&d_y, size * sizeof(double));
    hipMemcpy(d_y, y, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&d_cubed_diffs, (size - 1) * sizeof(double));
    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    compute_cubed_differences<<<blocks, threadsPerBlock>>>(d_y, d_cubed_diffs, size);
    double mean_cubed_diffs = compute_mean_cuda(d_cubed_diffs, size - 1);
    hipFree(d_y);
    hipFree(d_cubed_diffs);
    return mean_cubed_diffs;
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    double *y = NULL;
    int size = 0;
    FILE *fp = fopen("/mnt/c/Users/yashw/Downloads/test.txt", "rb");
    if (!fp) {
        fprintf(stderr, "Failed to open the file.\n");
        return 1;
    }

    double temp;
    while (fscanf(fp, "%lf", &temp) == 1) {
        size++;
    }

    fseek(fp, 0, SEEK_SET);
    y = (double *)malloc(size * sizeof(double));
    if (!y) {
        fprintf(stderr, "Failed to allocate memory.\n");
        fclose(fp);
        return 1;
    }

    for (int i = 0; i < size; i++) {
        if (fscanf(fp, "%lf", &y[i]) != 1) {
            fprintf(stderr, "Failed to read data from file.\n");
            free(y);
            fclose(fp);
            return 1;
        }
    }

    fclose(fp);
    int firstMinIndex = CO_FirstMin_ac_cuda(y, size);
    printf("CO_First_min: %d\n", firstMinIndex);

    double *autocorr_d = cuda_co_autocorrs(y, size);
    float result = CO_f1ecac_CUDA(autocorr_d, size);
    float result_2 = CO_trev_1_num_cuda(y, size);
    printf("CO_F1ecac : %f\n", result);
    printf("CO_trev_num1 : %.14f\n", result_2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(y);
    return 0;
}
